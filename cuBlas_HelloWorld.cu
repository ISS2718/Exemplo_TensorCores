
#include <hip/hip_runtime.h>
#include <stdio.h> // Inclui a biblioteca padrão de entrada e saída
#include <hipblas.h> // Inclui a biblioteca cuBLAS para operações de álgebra linear
#include <hiprand/hiprand_kernel.h> // Inclui a biblioteca CURAND para geração de números aleatórios


#define TAMANHO_MATRIZ 8 // Multiplos de 8 para usar Tensor Cores em FP64

// Função para inicializar uma matriz identidade
__global__ void init_identity(double *a, int n) {
    int idx = threadIdx.x;
    if (idx < n) {
        a[idx*n + idx] = 1.0;
    }
}

// Função para inicializar uma matriz de uns
__global__ void init_unit(double *b, int n) {
    int idx = threadIdx.x;
    if (idx < n) {
        for (int j = 0; j < n; j++) {
            b[idx*n + j] = 1.0;
        }
    }
}

// Função para inicializar uma matriz com números aleatórios
__global__ void init_random(double *a, int n) {
    int idx = threadIdx.x;
    hiprandState state;
    hiprand_init(1234, idx, 0, &state);

    if (idx < n) {
        for (int j = 0; j < n; j++) {
            a[idx*n + j] = hiprand_uniform_double(&state);
        }
    }
}

int main(int argc, char** argv) { // Função principal
    hipblasHandle_t handle; // Declara um handle para a biblioteca cuBLAS

    const int N = TAMANHO_MATRIZ; // Define o tamanho da matriz (N x N)
    double *d_A, *d_B, *d_C, *d_D, *d_E; // Declara ponteiros para as matrizes A, B, C, D, E na memória do dispositivo
    
    // C = alpha * (A * B) + beta * (A * B)
    const double alpha = 1.0; // Define o valor de alpha para a operação de multiplicação de matrizes
    const double beta = 0.0; // Define o valor de beta para a operação de multiplicação de matrizes

    hipMalloc((void**)&d_A, N * N * sizeof(d_A[0])); // Aloca memória na GPU para a matriz A
    hipMalloc((void**)&d_B, N * N * sizeof(d_B[0])); // Aloca memória na GPU para a matriz B
    hipMalloc((void**)&d_C, N * N * sizeof(d_C[0])); // Aloca memória na GPU para a matriz C
    hipMalloc((void**)&d_D, N * N * sizeof(d_D[0])); // Aloca memória na GPU para a matriz D
    hipMalloc((void**)&d_E, N * N * sizeof(d_E[0])); // Aloca memória na GPU para a matriz E

    // Inicializa as matrizes A e B na GPU
    init_identity<<<1, N>>>(d_A, N);
    init_unit<<<1, N>>>(d_B, N);
    init_random<<<1, N>>>(d_C, N);

    printf("Inicia Matrizes!\n"); // Imprime uma mensagem de inicio
    hipblasCreate(&handle); // Inicializa o handle cuBLAS
    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH); // Configura o modo de matemática para usar Tensor Cores
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_D, N); // Realiza a multiplicação de matrizes na GPU
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_C, N, &beta, d_E, N); // Realiza a multiplicação de matrizes na GPU

    hipDeviceSynchronize(); // Espera a GPU terminar de executar a multiplicação de matrizes
    printf("Multiplicacao de matrizes realizada com sucesso!\n"); // Imprime uma mensagem de sucesso

    // Copia a matriz resultante para a memória do host
    double* h_D = (double*)malloc(N * N * sizeof(double));
    hipMemcpy(h_D, d_D, N * N * sizeof(double), hipMemcpyDeviceToHost);

    double* h_E = (double*)malloc(N * N * sizeof(double));
    hipMemcpy(h_E, d_E, N * N * sizeof(double), hipMemcpyDeviceToHost);

    // Imprime a matriz resultante
    printf("Matriz D:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%f ", h_D[i * N + j]);
        }
        printf("\n");
    }

    // Imprime a matriz resultante
    printf("\nMatriz E:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%f ", h_E[i * N + j]);
        }
        printf("\n");
    }

    // Libera a memória
    free(h_D);
    free(h_E);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipDeviceReset(); // Reseta o dispositivo CUDA
    return 0; // Retorna 0 indicando que o programa terminou com sucesso
}
